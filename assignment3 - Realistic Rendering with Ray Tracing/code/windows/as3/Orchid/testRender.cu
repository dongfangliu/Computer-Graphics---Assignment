#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include ""
__global__ void renderRay(int samples, Camera * _camera, Vector3d * _pixelBuffer,Scene * _scene) {
	int x = blockIdx.x; int y = blockIdx.y; int a = threadIdx.x; int width = _camera->imageW();	int height = _camera->imageH();
	fprintf(stderr, "\rRendering (%i samples): %.2f%% ", samples, (double)blockIdx.y / height * 100);
	unsigned short Xi[3] = { 0,0,y*y*y };
	Ray ray = _camera->get_ray(blockIdx.x, blockIdx.y, a> 0, Xi);
	_pixelBuffer[(y)*width + x] += _scene->trace_ray(ray, 0, Xi);
}